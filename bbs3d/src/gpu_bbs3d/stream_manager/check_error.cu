#include <gpu_bbs3d/stream_manager/check_error.cuh>

namespace gpu {

void CUDACheckError::operator<<(hipError_t error) const {
  if (error == hipSuccess) {
    return;
  }

  const std::string error_name = hipGetErrorName(error);
  const std::string error_string = hipGetErrorString(error);

  std::cerr << "warning: " << error_name << std::endl;
  std::cerr << "       : " << error_string << std::endl;
}

CUDACheckError check_error;

}  // namespace gpu