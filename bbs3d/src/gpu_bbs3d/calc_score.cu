#include "hip/hip_runtime.h"
#include <gpu_bbs3d/bbs3d.cuh>
#include <gpu_bbs3d/voxelmaps.cuh>
#include <gpu_bbs3d/stream_manager/check_error.cuh>

namespace gpu {
__global__ void calc_scores_kernel(
  const thrust::device_ptr<Eigen::Vector4i*> multi_buckets_ptrs,
  const thrust::device_ptr<VoxelMapInfo> voxelmap_info_ptr,
  thrust::device_ptr<DiscreteTransformation> trans_ptr,
  size_t index_size,
  const thrust::device_ptr<Eigen::Vector3f> points_ptr,
  size_t num_points) {
  const size_t pose_index = threadIdx.x + blockIdx.x * blockDim.x;
  if (pose_index > index_size) {
    return;
  }

  DiscreteTransformation& trans = *thrust::raw_pointer_cast(trans_ptr + pose_index);
  const VoxelMapInfo& voxelmap_info = *thrust::raw_pointer_cast(voxelmap_info_ptr + trans.level);
  const Eigen::Vector4i* buckets = thrust::raw_pointer_cast(multi_buckets_ptrs)[trans.level];

  int score = 0;
  for (size_t i = 0; i < num_points; i++) {
    const Eigen::Vector3f& point = thrust::raw_pointer_cast(points_ptr)[i];

    const Eigen::Vector3f translation(trans.x, trans.y, trans.z);
    Eigen::Matrix3f rotation;
    rotation = Eigen::AngleAxisf(trans.yaw, Eigen::Vector3f::UnitZ()) * Eigen::AngleAxisf(trans.pitch, Eigen::Vector3f::UnitY()) *
               Eigen::AngleAxisf(trans.roll, Eigen::Vector3f::UnitX());
    const Eigen::Vector3f transed_point = rotation * point + translation;

    // coord to hash
    const Eigen::Vector3i coord = (transed_point.array() * voxelmap_info.inv_res).floor().cast<int>();
    const std::uint32_t hash = (coord[0] * 73856093) ^ (coord[1] * 19349669) ^ (coord[2] * 83492791);

    for (int j = 0; j < voxelmap_info.max_bucket_scan_count; j++) {
      const std::uint32_t bucket_index = (hash + j) % voxelmap_info.num_buckets;
      const Eigen::Vector4i bucket = buckets[bucket_index];

      if (bucket.x() != coord.x() || bucket.y() != coord.y() || bucket.z() != coord.z()) {
        continue;
      }

      if (bucket.w() == 1) {
        score++;
      }
    }
  }
  trans.score = score;
}

std::vector<DiscreteTransformation> BBS3D::calc_scores(const std::vector<DiscreteTransformation>& h_transset) {
  size_t transset_size = h_transset.size();
  thrust::device_vector<DiscreteTransformation> d_transset(transset_size);
  check_error << hipMemcpyAsync(
    thrust::raw_pointer_cast(d_transset.data()),
    h_transset.data(),
    sizeof(DiscreteTransformation) * transset_size,
    hipMemcpyHostToDevice,
    stream);

  const size_t block_size = 32;
  const size_t num_blocks = (transset_size + (block_size - 1)) / block_size;

  calc_scores_kernel<<<num_blocks, block_size, 0, stream>>>(
    voxelmaps_ptr_->d_multi_buckets_ptrs_.data(),
    voxelmaps_ptr_->d_voxelmaps_info_.data(),
    d_transset.data(),
    transset_size - 1,
    d_src_points_.data(),
    src_size_);

  std::vector<DiscreteTransformation> h_output(transset_size);
  check_error << hipMemcpyAsync(
    h_output.data(),
    thrust::raw_pointer_cast(d_transset.data()),
    sizeof(DiscreteTransformation) * transset_size,
    hipMemcpyDeviceToHost,
    stream);

  check_error << hipStreamSynchronize(stream);
  return h_output;
}
}  // namespace gpu